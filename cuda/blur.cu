#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
extern "C" {
#include "ppmFile.h"
}

// kernel function that blurs image based on block IDs.
__global__ void blur(int *d_width, int *d_height, int *d_radius, unsigned char *d_input, unsigned char*d_output){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    int offset;

    //for loop to loop through all the channels in a thread. 
    for(int channel = 0; channel < 3; channel++){
        int temp = 0;
        int num = 0;

        //nested for loop to go through all the pixels within the radius.
        for(int y = j - (*d_radius); y <= j + (*d_radius); y++)
        {
            for(int x = i - (*d_radius); x < i + (*d_radius); x++){
                
                if(x >=0 && x < *d_width && y>=0 && y < *d_height){
                    offset = (y * (*d_width) + x) * 3 + channel; //setting the offset of the current pixels to blur the image within radius.
                    temp += d_input[offset];
                    num++;
                }
            }
        }
        //averaging the pixel values.
        temp = temp / num;
        offset = (j * (*d_width) + i) * 3 + channel;
        d_output[offset] = temp;
    }

}

int main (int argc, char *argv[]){
    double time_DTH, time_allocateHTD,time_kernel;
    clock_t begin_allocateHTD, end_allocateHTD, begin_kernel, end_kernel, begin_DTH, end_DTH;
    
    
    
    //Host variable
    //width heigth of image and radius of the blur filter.
    int width, height, radius;
    //input image and output image struct. defined in ppmFile.
    Image *inImage, *outImage;
    //data of input image.
    unsigned char *data;

    //Device variable
    //input image data
    unsigned char *d_input;
    //output image data
    unsigned char *d_output;
    //width, height and radius passed to kernel
    int *d_width, *d_height, *d_radius;

    //unsigned char *output = (unsigned char *)malloc(sizeof(unsigned char*) * image_size);

    if(argc != 4){
        printf("Incorrect input argument should include radius, input file and output file.\n");
        return 0;
    }

    //initializing values.
    radius = atoi(argv[1]);
    inImage = ImageRead(argv[2]);
    width = inImage->width;
    height = inImage->height;
    data = inImage->data;

    //check the values of the images.
    printf("Using image: %s, width: %d, height: %d, blur radius: %d\n",argv[2],width,height,radius);


    //Grids based on size of the block 32 * 32
    dim3 blockD(32,32);
    dim3 gridD((width + blockD.x - 1)/blockD.x, (height + blockD.y - 1)/blockD.y);

    //size of image pixels. 3 is number of channels. 
    int image_size = width * height * 3;

    begin_allocateHTD = clock();
    //allocate memory for GPU
    hipMalloc((void**)&d_input, sizeof(unsigned char*) * image_size);
    hipMalloc((void**)&d_output, sizeof(unsigned char*) * image_size);
    hipMalloc((void**)&d_radius, sizeof(int*));
    hipMalloc((void**)&d_height, sizeof(int*));
    hipMalloc((void**)&d_width, sizeof(int*));

    //copy values to GPU
    //HostToDevice
    hipMemcpy(d_input, data, image_size, hipMemcpyHostToDevice);
    hipMemcpy(d_width, &width, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_height, &height, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_radius, &radius, sizeof(int), hipMemcpyHostToDevice);

    end_allocateHTD = clock();

    begin_kernel = clock();
    //call blur kernel for GPU execution.
    blur<<<gridD, blockD>>>(d_width, d_height, d_radius, d_input,d_output);

    end_kernel = clock();

    
    //create new image and clear the image to copy blurred image from the device.
    outImage = ImageCreate(width,height);
    ImageClear(outImage,255,255,255);

    
    begin_DTH = clock();
    hipDeviceSynchronize();

    //copy blured out image from gpu.
    //Device to Host
    hipMemcpy(outImage->data, d_output, image_size, hipMemcpyDeviceToHost);
    
    end_DTH = clock();
    //write blurred image into the file name passed as argument.
    ImageWrite(outImage, argv[3]);

    time_allocateHTD = (double)(end_allocateHTD-begin_allocateHTD) / CLOCKS_PER_SEC;
    printf("Allocation and Host to Device Time: %e s\n", time_allocateHTD);

    time_kernel = (double)(end_kernel-begin_kernel) / CLOCKS_PER_SEC;
    printf("Kernel Time: %e s\n", time_kernel);

    time_DTH = (double)(end_DTH-begin_DTH) / CLOCKS_PER_SEC;
    printf("Device to Host Time: %e s\n", time_DTH);

    printf("Total Time : %e s\n",time_allocateHTD + time_kernel + time_DTH);
    //free memory
    free(inImage->data);
    free(outImage->data);
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_width);
    hipFree(d_height);
    hipFree(d_radius);

    return 0;


}